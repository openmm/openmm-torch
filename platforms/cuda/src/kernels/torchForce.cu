#include "hip/hip_runtime.h"
extern "C" __global__
void copyInputs(real* __restrict__ posTensor, real* __restrict__ boxTensor, const real4* __restrict__ posq,
        int* __restrict__ atomIndex, int numAtoms, real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ) {
    for (int atom = blockIdx.x*blockDim.x+threadIdx.x; atom < numAtoms; atom += blockDim.x*gridDim.x) {
        int index = atomIndex[atom];
        real4 pos = posq[atom];
        posTensor[3*index] = pos.x;
        posTensor[3*index+1] = pos.y;
        posTensor[3*index+2] = pos.z;
    }
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        boxTensor[0] = periodicBoxVecX.x;
        boxTensor[1] = periodicBoxVecX.y;
        boxTensor[2] = periodicBoxVecX.z;
        boxTensor[3] = periodicBoxVecY.x;
        boxTensor[4] = periodicBoxVecY.y;
        boxTensor[5] = periodicBoxVecY.z;
        boxTensor[6] = periodicBoxVecZ.x;
        boxTensor[7] = periodicBoxVecZ.y;
        boxTensor[8] = periodicBoxVecZ.z;
    }
}

extern "C" __global__
void addForces(const real* __restrict__ grads, long long* __restrict__ forceBuffers, int* __restrict__ atomIndex, int numAtoms, int paddedNumAtoms, int forceSign) {
    for (int atom = blockIdx.x*blockDim.x+threadIdx.x; atom < numAtoms; atom += blockDim.x*gridDim.x) {
        int index = atomIndex[atom];
        forceBuffers[atom] += (long long) (forceSign*grads[3*index]*0x100000000);
        forceBuffers[atom+paddedNumAtoms] += (long long) (forceSign*grads[3*index+1]*0x100000000);
        forceBuffers[atom+2*paddedNumAtoms] += (long long) (forceSign*grads[3*index+2]*0x100000000);
    }
}

